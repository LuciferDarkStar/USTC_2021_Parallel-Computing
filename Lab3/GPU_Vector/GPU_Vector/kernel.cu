#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include <fstream>
#include <iomanip>
#include <stdio.h>

// Kernal:
__global__ void MatrixMultiply(double *a, double * b, double *c, int N) {
	int tx = threadIdx.x + blockIdx.x * blockDim.x;
	if (tx < N) {
		double sum = 0;
		for (int k = 0; k < N; ++k) {
			sum += a[tx * N + k] * b[k];
		}
		c[tx] = sum;
	}
}

hipError_t matrixMultiplyWithCuda(double *a, double *b, double *c, size_t size);

int main()
{
	std::ifstream in("data.txt");
	int N;
	in >> N;//矩阵阶数
	if (in.fail()) {
		printf("错误①！\n");
	}
	else {
		printf("成功！\n");
	}
	// host initial
	double *a = new double[N * N];
	double *b = new double[N];
	double *c = new double[N];

	// read 
	for (int i = 0; i < N; ++i)
		for (int j = 0; j < N; ++j) in >> a[i * N + j];

	for (int i = 0; i < N; ++i) in >> b[i];

	hipError_t cudaStatus = matrixMultiplyWithCuda(a, b, c, N);

	std::ofstream out("output.txt");
	for (int i = 0; i < N; ++i) {
		out << std::setiosflags(std::ios::fixed) << c[i] << " ";
		out << std::endl;
	}
	cudaStatus = hipDeviceReset();

	// host free 
	delete[] a;
	delete[] b;
	delete[] c;
	return 0;
}
hipError_t matrixMultiplyWithCuda(double *a, double *b, double *c, size_t N) {
	double *dev_a = 0;
	double *dev_b = 0;
	double *dev_c = 0;
	hipError_t cudaStatus;
	cudaStatus = hipMalloc((void**)&dev_a, N * N * sizeof(double));
	cudaStatus = hipMalloc((void**)&dev_b, N * sizeof(double));
	cudaStatus = hipMalloc((void**)&dev_c, N * sizeof(double));
	cudaStatus = hipMemcpy(dev_a, a, N * N * sizeof(double), hipMemcpyHostToDevice);
	cudaStatus = hipMemcpy(dev_b, b, N * sizeof(double), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		printf("错误②！\n");
		goto Error;
	}
	// kernal invocation 
	dim3 threadPerBlock(500, 1, 1);
	dim3 numBlocks(N / threadPerBlock.x + 1, 1, 1);
	MatrixMultiply <<<numBlocks, threadPerBlock >>> (dev_a, dev_b, dev_c, N);
	if (cudaStatus != hipSuccess) {
		printf("计算错误\n");
		goto Error;
	}
	cudaStatus = hipMemcpy(c, dev_c, N * sizeof(double), hipMemcpyDeviceToHost);
Error:
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);
	return cudaStatus;
}
