﻿#include "hip/hip_runtime.h"


#include <stdio.h>

#include<string.h>
#include<ctime>

#define BLOCK_NUM 4   // 块数量
#define THREAD_NUM 2 // 每个块中的线程数
#define R_SIZE (BLOCK_NUM * THREAD_NUM) // 矩阵行列数
#define M_SIZE (R_SIZE * R_SIZE) //矩阵规模

__global__ void mat_mul(int* mat1, int* mat2, int* result)
{
	const int bid = blockIdx.x; //块 id
	const int tid = threadIdx.x; //进程 id
	// 每个线程计算一行
	const int row = bid * THREAD_NUM + tid; //计算当前进程所需计算的行数
	for (int c = 0; c < R_SIZE; c++)
	{
		for (int n = 0; n < R_SIZE; n++)
		{
			result[row * R_SIZE + c] += mat1[row * R_SIZE + n] * mat2[n * R_SIZE + c];
		}
	}
}

int main(int argc, char* argv[])
{
	int* mat1, *mat2, *result;
	int* g_mat1, *g_mat2, *g_mat_result;
	double time_pc, time_normal;

	clock_t startTime, endTime;

	// 用一位数组表示二维矩阵
	mat1 = (int*)malloc(M_SIZE * sizeof(int));
	mat2 = (int*)malloc(M_SIZE * sizeof(int));
	result = (int*)malloc(M_SIZE * sizeof(int));

	// initialize
	for (int i = 0; i < M_SIZE; i++)
	{
		mat1[i] = rand() % 10;
		mat2[i] = rand() % 10;
		result[i] = 0;
	}

	printf("矩阵 1 为：\n");
	for (int i = 0; i < M_SIZE; i++)
		if((i + 1) % R_SIZE == 0)
			printf("%d\n", mat1[i]);
		else
			printf("%d ", mat1[i]);

	printf("\n矩阵 2 为：\n");
	for (int i = 0; i < M_SIZE; i++)
		if ((i + 1) % R_SIZE == 0)
			printf("%d\n", mat2[i]);
		else
			printf("%d ", mat2[i]);
	

	hipMalloc((void**)&g_mat1, sizeof(int) * M_SIZE);
	hipMalloc((void**)&g_mat2, sizeof(int) * M_SIZE);
	hipMalloc((void**)&g_mat_result, sizeof(int) * M_SIZE);

	hipMemcpy(g_mat1, mat1, sizeof(int) * M_SIZE, hipMemcpyHostToDevice);
	hipMemcpy(g_mat2, mat2, sizeof(int) * M_SIZE, hipMemcpyHostToDevice);

	/*并行方法*/
	startTime = clock();//计时开始

	mat_mul <<<BLOCK_NUM, THREAD_NUM >>> (g_mat1, g_mat2, g_mat_result);

	hipMemcpy(result, g_mat_result, sizeof(int) * M_SIZE, hipMemcpyDeviceToHost);

	endTime = clock();//计时结束
	time_pc = (double)(endTime - startTime) / CLOCKS_PER_SEC;
	printf("并行所用时间: %lf s\n", time_pc);


	/*串行方法*/
	startTime = clock();//计时开始

	for (int r = 0; r < R_SIZE; r++) {
		for (int c = 0; c < R_SIZE; c++) {
			for (int n = 0; n < R_SIZE; n++) {
				result[r * R_SIZE + c] += mat1[r * R_SIZE + n] * mat2[n * R_SIZE + c];
			}
		}
	}

	endTime = clock();//计时结束
	time_normal = (double)(endTime - startTime) / CLOCKS_PER_SEC;
	printf("串行所用时间: %lf s\n", time_normal);

	printf("加速比为：%lf\n", time_normal / time_pc);

	printf("\n二矩阵乘积为：\n");
	for (int i = 0; i < M_SIZE; i++)
		if ((i + 1) % R_SIZE == 0)
			printf("%d\n\n", result[i]);
		else
			printf("%d ", result[i]);
}